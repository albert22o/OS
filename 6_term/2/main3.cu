#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#include <chrono>
using namespace std;
typedef std::chrono::milliseconds ms;
typedef std::chrono::nanoseconds ns;

__global__ void vectorAdd(const float* a, const float* b, float* c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    for (long n = 10; n <= 100000000; n *= 10)
  {
    cout << endl
         << "n = " << n << endl;
    float elapsedTime;
    hipEvent_t start, stop;
    chrono::time_point<chrono::system_clock> start_chrono, end_chrono;

    float* d_a, * d_b, * d_c;
    hipMalloc((void**)&d_a, n * sizeof(float));
    hipMalloc((void**)&d_b, n * sizeof(float));
    hipMalloc((void**)&d_c, n * sizeof(float));

    float* h_a = new float[n];
    float* h_b = new float[n];
    for (int i = 0; i < n; ++i) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    hipMemcpy(d_a, h_a, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, n * sizeof(float), hipMemcpyHostToDevice);

    // Вычисляем количество блоков и нитей на блок
    int blockSize = 1024;
    int numBlocks = n;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    start_chrono = chrono::system_clock::now();
    vectorAdd <<< numBlocks, blockSize >>> (d_a, d_b, d_c, n);
    hipEventRecord(stop, 0);
    end_chrono = chrono::system_clock::now();

    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    float* h_c = new float[n];
    hipMemcpy(h_c, d_c, n * sizeof(float), hipMemcpyDeviceToHost);

    cout <<"CUDA Event time: "<< elapsedTime * 1000 << "ns" << endl
         <<"Chrono time: "<< chrono::duration_cast<ms>(end_chrono - start_chrono).count() << "ms"
        << endl << chrono::duration_cast<ns>(end_chrono - start_chrono).count() << "ns" << endl;


    delete[] h_a;
    delete[] h_b;
    delete[] h_c;
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

  }
}
