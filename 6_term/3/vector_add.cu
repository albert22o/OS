#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#include <chrono>
using namespace std;
typedef std::chrono::milliseconds ms;
typedef std::chrono::nanoseconds ns;

__global__ void vectorAdd(const float *a, const float *b, float *c, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        // Искусственно вызываем ошибку для некоторых нитей
        if (i % 100 == 0)
        {
            c[i] = a[i + 1000] + b[i]; // Обращение к элементу за пределами массива
        }
        else
        {
            c[i] = a[i] + b[i];
        }
    }
}

int main()
{
    for (long n = 10; n <= 100000000; n *= 10)
    {
        cout << endl
             << "n = " << n << endl;
        float elapsedTime;
        hipEvent_t start, stop;
        chrono::time_point<chrono::system_clock> start_chrono, end_chrono;

        float *d_a, *d_b, *d_c;
        hipMalloc((void **)&d_a, n * sizeof(float));
        hipMalloc((void **)&d_b, n * sizeof(float));
        hipMalloc((void **)&d_c, n * sizeof(float));

        float *h_a = new float[n];
        float *h_b = new float[n];
        for (int i = 0; i < n; ++i)
        {
            h_a[i] = i;
            h_b[i] = i * 2;
        }

        hipMemcpy(d_a, h_a, n * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_b, h_b, n * sizeof(float), hipMemcpyHostToDevice);

        int blockSize = 1024; // Количество нитей в блоке
        int numBlocks = n;    // Количество блоков

        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start, 0);
        start_chrono = chrono::system_clock::now();
        vectorAdd<<<numBlocks, blockSize>>>(d_a, d_b, d_c, n);

        // Проверка ошибок выполнения ядра
        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("Kernel execution failed: %s\n", hipGetErrorString(err));
        }

        hipDeviceSynchronize(); // Синхронизация для проверки ошибок
        hipEventRecord(stop, 0);
        end_chrono = chrono::system_clock::now();

        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop);

        float *h_c = new float[n];
        hipMemcpy(h_c, d_c, n * sizeof(float), hipMemcpyDeviceToHost);

        cout << "CUDA Event time: " << elapsedTime * 1000 << "ns" << endl
             << "Chrono time: " << chrono::duration_cast<ms>(end_chrono - start_chrono).count() << "ms"
             << endl
             << chrono::duration_cast<ns>(end_chrono - start_chrono).count() << "ns" << endl;

        delete[] h_a;
        delete[] h_b;
        delete[] h_c;
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
    }
    return 0;
}
