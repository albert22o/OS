#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <vector>
#include <hip/hip_runtime.h>

// === Константная память ===
__constant__ float d_center[3]; // x0, y0, z0
__constant__ float d_radius;    // R

// === Текстурная память ===
texture<float, 3, hipReadModeElementType> tex3DRef;

// === Ядро CUDA ===
__global__ void integrateOnSphereKernel(float *partialSums, int thetaSteps, int phiSteps)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int totalPoints = thetaSteps * phiSteps;
    if (tid >= totalPoints)
        return;

    int i = tid / phiSteps;
    int j = tid % phiSteps;

    float theta = M_PI * (i + 0.5f) / thetaSteps;
    float phi = 2.0f * M_PI * (j + 0.5f) / phiSteps;

    float x = d_center[0] + d_radius * sinf(theta) * cosf(phi);
    float y = d_center[1] + d_radius * sinf(theta) * sinf(phi);
    float z = d_center[2] + d_radius * cosf(theta);

    float val = tex3D(tex3DRef, x, y, z);

    float dtheta = M_PI / thetaSteps;
    float dphi = 2.0f * M_PI / phiSteps;
    float dS = d_radius * d_radius * sinf(theta) * dtheta * dphi;

    partialSums[tid] = val * dS;
}

// === Функция интегрирования с замером времени ===
float integrateOnSphere(float *h_volume, int nx, int ny, int nz,
                        float x0, float y0, float z0, float R,
                        int thetaSteps, int phiSteps)
{
    // Подготовка 3D текстурного массива
    hipExtent volumeSize = make_hipExtent(nx, ny, nz);
    hipArray *d_array;
    hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
    hipMalloc3DArray(&d_array, &desc, volumeSize);

    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr = make_hipPitchedPtr(h_volume, nx * sizeof(float), nx, ny);
    copyParams.dstArray = d_array;
    copyParams.extent = volumeSize;
    copyParams.kind = hipMemcpyHostToDevice;
    hipMemcpy3D(&copyParams);

    // Привязка текстуры
    tex3DRef.normalized = true;                 // используем реальные координаты
    tex3DRef.filterMode = hipFilterModeLinear; // отключаем линейную интерполяцию
    tex3DRef.addressMode[0] = hipAddressModeClamp;
    tex3DRef.addressMode[1] = hipAddressModeClamp;
    tex3DRef.addressMode[2] = hipAddressModeClamp;
    hipBindTextureToArray(tex3DRef, d_array, desc);

    // Копирование параметров в константную память
    float h_center[3] = {x0, y0, z0};
    hipMemcpyToSymbol(HIP_SYMBOL(d_center), h_center, sizeof(float) * 3);
    hipMemcpyToSymbol(HIP_SYMBOL(d_radius), &R, sizeof(float));

    // Настройка ядра
    int totalPoints = thetaSteps * phiSteps;
    float *d_partialSums;
    hipMalloc(&d_partialSums, sizeof(float) * totalPoints);

    int blockSize = 256;
    int gridSize = (totalPoints + blockSize - 1) / blockSize;

    // === Замер времени выполнения ядра ===
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    integrateOnSphereKernel<<<gridSize, blockSize>>>(d_partialSums, thetaSteps, phiSteps);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "GPU kernel execution time: " << milliseconds << " ms" << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Получение результата
    std::vector<float> h_partialSums(totalPoints);
    hipMemcpy(h_partialSums.data(), d_partialSums, sizeof(float) * totalPoints, hipMemcpyDeviceToHost);

    float result = 0.0f;
    for (float val : h_partialSums)
        result += val;

    // Очистка
    hipUnbindTexture(tex3DRef);
    hipFreeArray(d_array);
    hipFree(d_partialSums);

    return result;
}

// === Создание объёма: f(x,y,z) = 1 ===
void generateConstantVolume(std::vector<float> &volume, int nx, int ny, int nz, float value)
{
    volume.resize(nx * ny * nz, value);
}

// === main ===
int main()
{
    const int nx = 64, ny = 64, nz = 64;

    std::vector<float> volume;
    generateConstantVolume(volume, nx, ny, nz, 1.0f); // f(x, y, z) = 1

    float x0 = nx / 2.0f;
    float y0 = ny / 2.0f;
    float z0 = nz / 2.0f;
    float R = 10.0f;

    int thetaSteps = 180;
    int phiSteps = 360;

    float integral = integrateOnSphere(volume.data(), nx, ny, nz, x0, y0, z0, R, thetaSteps, phiSteps);

    std::cout << "--- TEXTURE OPTIMIZED ---" << std::endl;
    std::cout << "Computed integral: " << integral << std::endl;
    std::cout << "Expected (4πR²): " << 4.0 * M_PI * R * R << std::endl;

    return 0;
}
